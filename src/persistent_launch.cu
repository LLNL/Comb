//////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2018, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory
//
// Created by Jason Burmark, burmark1@llnl.gov
// LLNL-CODE-758885
//
// All rights reserved.
//
// This file is part of Comb.
//
// For details, see https://github.com/LLNL/Comb
// Please also see the LICENSE file for MIT license.
//////////////////////////////////////////////////////////////////////////////

#include "persistent_launch.cuh"

#include <hip/hip_cooperative_groups.h>

namespace cuda {

namespace persistent_launch {

namespace detail {

// Launches a batch kernel and cycles to next buffer
void launch(::detail::MultiBuffer& mb, hipStream_t stream)
{
   // NVTX_RANGE_COLOR(NVTX_CYAN)
   if (!getLaunched()) {

      // ensure other thread/device done reading buffer before launch
      while(!mb.cur_buffer_empty());
      // get empty unused buffer
      ::detail::MultiBuffer::buffer_type* device_buffer = mb.get_buffer();

      int blocks_cutoff = 1;
      // TODO decide cutoff below which all blocks read pinned memory directly
      // if (::detail::cuda::get_cuda_arch() >= 600) {
      //    // pascal or newer
      //    blocks_cutoff = 1;
      // } else {
      //    blocks_cutoff = 1;
      // }

      int blocksize = 1024;
      // TODO decide blocksize in a smart way

      int num_blocks = ::detail::cuda::get_num_sm();

      void* func = NULL;
      void* args[] = { (void*)&device_buffer };

      if (num_blocks < blocks_cutoff) {
         // don't use device cache
         if (get_batch_always_grid_sync()) {
            func = (void*)&::detail::block_read_device<::detail::MultiBuffer::shared_buffer_type>;
         } else {
            func = (void*)&::detail::block_read_device_few_grid_sync<::detail::MultiBuffer::shared_buffer_type>;
         }
      } else {
         // use device cache
         if (get_batch_always_grid_sync()) {
            func = (void*)&::detail::block_read_device<::detail::MultiBuffer::shared_buffer_type>;
         } else {
            func = (void*)&::detail::block_read_device_few_grid_sync<::detail::MultiBuffer::shared_buffer_type>;
         }
      }
      cudaCheck(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), num_blocks, blocksize,
                                            args, 0, stream));
      getLaunched() = true;
   }
}

void stop(::detail::MultiBuffer& mb, hipStream_t stream)
{
   if (getLaunched()) {
     mb.done_packing();
     getLaunched() = false;
   }
}

} // namespace detail

// Start the current batch (launches kernel)
void force_launch(hipStream_t stream)
{
   // NVTX_RANGE_COLOR(NVTX_CYAN)
   if (!detail::getLaunched()) {
      detail::launch(detail::getMultiBuffer(), stream);
   }
}

// Ensure current batch launched (does nothing)
void force_stop(hipStream_t stream)
{
   if (detail::getLaunched()) {
      detail::stop(detail::getMultiBuffer(), stream);
   }
}

// Wait for all batches to finish running
void synchronize(hipStream_t stream)
{
   // NVTX_RANGE_COLOR(NVTX_CYAN)
   force_stop(stream);

   // perform synchronization
   cudaCheck(hipDeviceSynchronize());
}

} // namespace persistent_launch

} // namespace cuda

