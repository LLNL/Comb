#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <cctype>

#include <mpi.h>

#include "memory.cuh"
#include "for_all.cuh"
#include "profiling.cuh"
#include "MeshInfo.cuh"
#include "MeshData.cuh"
#include "comm.cuh"
#include "CommFactory.cuh"

namespace detail {

  struct set_n1 {
     DataT* data;
     set_n1(DataT* data_) : data(data_) {}
     HOST DEVICE
     void operator()(IdxT i, IdxT) const {
       IdxT zone = i;
       //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
       data[zone] = -1.0;
     }
  };

  struct set_1 {
     IdxT ilen, ijlen;
     DataT* data;
     set_1(IdxT ilen_, IdxT ijlen_, DataT* data_) : ilen(ilen_), ijlen(ijlen_), data(data_) {}
     HOST DEVICE
     void operator()(IdxT k, IdxT j, IdxT i, IdxT idx) const {
       IdxT zone = i + j * ilen + k * ijlen;
       //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
       data[zone] = 1.0;
     }
  };

  struct reset_1 {
     IdxT ilen, ijlen;
     DataT* data;
     IdxT imin, jmin, kmin;
     IdxT imax, jmax, kmax;
     reset_1(IdxT ilen_, IdxT ijlen_, DataT* data_, IdxT imin_, IdxT jmin_, IdxT kmin_, IdxT imax_, IdxT jmax_, IdxT kmax_)
       : ilen(ilen_), ijlen(ijlen_), data(data_)
       , imin(imin_), jmin(jmin_), kmin(kmin_)
       , imax(imax_), jmax(jmax_), kmax(kmax_)
     {}
     HOST DEVICE
     void operator()(IdxT k, IdxT j, IdxT i, IdxT idx) const {
       IdxT zone = i + j * ilen + k * ijlen;
       DataT expected, found, next;
       if (k >= kmin && k < kmax &&
           j >= jmin && j < jmax &&
           i >= imin && i < imax) {
         expected = 1.0; found = data[zone]; next = 1.0;
       } else {
         expected = 0.0; found = data[zone]; next = -1.0;
       }
       //if (found != expected) FPRINTF(stdout, "zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
       //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
       data[zone] = next;
     }
  };

} // namespace detail

template < typename pol_loop, typename pol_many, typename pol_few >
void do_cycles(CommInfo& comm_info, MeshInfo& info, IdxT num_vars, IdxT ncycles, Allocator& aloc_mesh, Allocator& aloc_many, Allocator& aloc_few, Timer& tm)
{
    tm.clear();

    char rname[1024] = ""; snprintf(rname, 1024, "Buffers %s %s %s %s", pol_many::name, aloc_many.name(), pol_few::name, aloc_few.name());
    char test_name[1024] = ""; snprintf(test_name, 1024, "Mesh %s %s %s", pol_loop::name, aloc_mesh.name(), rname);
    FPRINTF(stdout, "Starting test %s\n", test_name);

    Range r0(rname, Range::orange);
    
    comm_info.barrier();

    tm.start("start-up");

    std::vector<MeshData> vars;
    vars.reserve(num_vars);
    
    Comm<pol_many, pol_few> comm(comm_info, aloc_many, aloc_few);
    
    {
      CommFactory factory(comm_info);
      
      for (IdxT i = 0; i < num_vars; ++i) {
    
        vars.push_back(MeshData(info, aloc_mesh));
      
        vars[i].allocate();
      
        DataT* data = vars[i].data();
        IdxT totallen = info.totallen;

        for_all(pol_loop{}, 0, totallen,
                            detail::set_n1(data));

        factory.add_var(vars[i]);
      
        synchronize(pol_loop{});
      }
      
      factory.populate(comm);
    }

    tm.stop();
    
    { // test comm

      Range r1("test comm", Range::magenta);

      IdxT imin = info.min[0];
      IdxT jmin = info.min[1];
      IdxT kmin = info.min[2];
      IdxT imax = info.max[0];
      IdxT jmax = info.max[1];
      IdxT kmax = info.max[2];
      IdxT ilen = info.len[0];
      IdxT jlen = info.len[1];
      IdxT klen = info.len[2];
      IdxT iglobal_offset = info.global_offset[0];
      IdxT jglobal_offset = info.global_offset[1];
      IdxT kglobal_offset = info.global_offset[2];
      IdxT ilen_global = info.global.sizes[0];
      IdxT jlen_global = info.global.sizes[1];
      IdxT klen_global = info.global.sizes[2];
      IdxT iperiodic = info.global.periodic[0];
      IdxT jperiodic = info.global.periodic[1];
      IdxT kperiodic = info.global.periodic[2];
      IdxT ijlen = info.stride[2];
      IdxT ijlen_global = ilen_global * jlen_global;
      
      
      Range r2("pre-comm", Range::red);
      // tm.start("pre-comm");

      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
        IdxT var_i = i;
      
        for_all_3d(pol_loop{}, 0, klen,
                               0, jlen,
                               0, ilen,
                               [=] HOST DEVICE (IdxT k, IdxT j, IdxT i, IdxT idx) {
          IdxT zone = i + j * ilen + k * ijlen;
          IdxT iglobal = i + iglobal_offset;
          if (iperiodic) {
            iglobal = iglobal % ilen_global;
            if (iglobal < 0) iglobal += ilen_global;
          }
          IdxT jglobal = j + jglobal_offset;
          if (jperiodic) {
            jglobal = jglobal % jlen_global;
            if (jglobal < 0) jglobal += jlen_global;
          }
          IdxT kglobal = k + kglobal_offset;
          if (kperiodic) {
            kglobal = kglobal % klen_global;
            if (kglobal < 0) kglobal += klen_global;
          }
          IdxT zone_global = iglobal + jglobal * ilen_global + kglobal * ijlen_global;
          DataT expected, found, next;
          if (k >= kmin && k < kmax &&
              j >= jmin && j < jmax &&
              i >= imin && i < imax) {
            next = zone_global + var_i;
          } else if (iglobal < 0 || iglobal >= ilen_global ||
                     jglobal < 0 || jglobal >= jlen_global ||
                     kglobal < 0 || kglobal >= klen_global) {
            next = -zone_global - var_i;
          } else {
            next = -zone_global - var_i;
          }
          data[zone] = next;
        });
      }
      
      synchronize(pol_loop{});

      // tm.stop();
      r2.restart("post-recv", Range::pink);
      // tm.start("post-recv");
      
      comm.postRecv();

      // tm.stop();
      r2.restart("post-send", Range::pink);
      // tm.start("post-send");

      comm.postSend();
      
      // tm.stop();
      r2.stop();
      
      
      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
        IdxT var_i = i;
        
        for_all_3d(pol_loop{}, 0, klen,
                               0, jlen,
                               0, ilen,
                               [=] HOST DEVICE (IdxT k, IdxT j, IdxT i, IdxT idx) {
          IdxT zone = i + j * ilen + k * ijlen;
          IdxT iglobal = i + iglobal_offset;
          if (iperiodic) {
            iglobal = iglobal % ilen_global;
            if (iglobal < 0) iglobal += ilen_global;
          }
          IdxT jglobal = j + jglobal_offset;
          if (jperiodic) {
            jglobal = jglobal % jlen_global;
            if (jglobal < 0) jglobal += jlen_global;
          }
          IdxT kglobal = k + kglobal_offset;
          if (kperiodic) {
            kglobal = kglobal % klen_global;
            if (kglobal < 0) kglobal += klen_global;
          }
          IdxT zone_global = iglobal + jglobal * ilen_global + kglobal * ijlen_global;
          DataT expected, found, next;
          if (k >= kmin && k < kmax &&
              j >= jmin && j < jmax &&
              i >= imin && i < imax) {
            expected = zone_global + var_i;  found = data[zone]; next = -1.0;
          } else if (iglobal < 0 || iglobal >= ilen_global ||
                     jglobal < 0 || jglobal >= jlen_global ||
                     kglobal < 0 || kglobal >= klen_global) {
            expected = -zone_global - var_i; found = data[zone]; next = -zone_global - var_i;
          } else {
            expected = -zone_global - var_i; found = data[zone]; next = 1.0;
          }
          //if (found != expected) FPRINTF(stdout, "zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
          //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
          assert(found == expected);
          data[zone] = next;
        });
      }
      

      r2.start("wait-recv", Range::pink);
      // tm.start("wait-recv");

      comm.waitRecv();

      // tm.stop();
      r2.restart("wait-send", Range::pink);
      // tm.start("wait-send");

      comm.waitSend();

      // tm.stop();
      r2.restart("post-comm", Range::red);
      // tm.start("post-comm");

      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
        IdxT var_i = i;
        
        for_all_3d(pol_loop{}, 0, klen,
                               0, jlen,
                               0, ilen,
                               [=] HOST DEVICE (IdxT k, IdxT j, IdxT i, IdxT idx) {
          IdxT zone = i + j * ilen + k * ijlen;
          IdxT iglobal = i + iglobal_offset;
          if (iperiodic) {
            iglobal = iglobal % ilen_global;
            if (iglobal < 0) iglobal += ilen_global;
          }
          IdxT jglobal = j + jglobal_offset;
          if (jperiodic) {
            jglobal = jglobal % jlen_global;
            if (jglobal < 0) jglobal += jlen_global;
          }
          IdxT kglobal = k + kglobal_offset;
          if (kperiodic) {
            kglobal = kglobal % klen_global;
            if (kglobal < 0) kglobal += klen_global;
          }
          IdxT zone_global = iglobal + jglobal * ilen_global + kglobal * ijlen_global;
          DataT expected, found, next;
          if (k >= kmin && k < kmax &&
              j >= jmin && j < jmax &&
              i >= imin && i < imax) {
            expected = -1.0;                 found = data[zone]; next = 1.0;
          } else if (iglobal < 0 || iglobal >= ilen_global ||
                     jglobal < 0 || jglobal >= jlen_global ||
                     kglobal < 0 || kglobal >= klen_global) {
            expected = -zone_global - var_i; found = data[zone]; next = -1.0;
          } else {
            expected = zone_global + var_i;  found = data[zone]; next = -1.0;
          }
          //if (found != expected) FPRINTF(stdout, "zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
          //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
          assert(found == expected);
          data[zone] = next;
        });
      }
      
      synchronize(pol_loop{});

      // tm.stop();
      r2.stop();

    }

    for(IdxT cycle = 0; cycle < ncycles; cycle++) {

      Range r1("cycle", Range::yellow);

      IdxT imin = info.min[0];
      IdxT jmin = info.min[1];
      IdxT kmin = info.min[2];
      IdxT imax = info.max[0];
      IdxT jmax = info.max[1];
      IdxT kmax = info.max[2];
      IdxT ilen = info.len[0];
      IdxT jlen = info.len[1];
      IdxT klen = info.len[2];
      IdxT ijlen = info.stride[2];
      
      
      Range r2("pre-comm", Range::red);
      tm.start("pre-comm");

      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
      
        for_all_3d(pol_loop{}, kmin, kmax,
                               jmin, jmax,
                               imin, imax,
                               detail::set_1(ilen, ijlen, data));
      }
      
      synchronize(pol_loop{});

      tm.stop();
      r2.restart("post-recv", Range::pink);
      tm.start("post-recv");
      
      comm.postRecv();

      tm.stop();
      r2.restart("post-send", Range::pink);
      tm.start("post-send");

      comm.postSend();
      
      tm.stop();
      r2.stop();
      
      /*
      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
        
        for_all_3d(pol_loop{}, 0, klen,
                               0, jlen,
                               0, ilen,
                               [=] HOST DEVICE (IdxT k, IdxT j, IdxT i, IdxT idx) {
          IdxT zone = i + j * ilen + k * ijlen;
          DataT expected, found, next;
          if (k >= kmin && k < kmax &&
              j >= jmin && j < jmax &&
              i >= imin && i < imax) {
            expected = 1.0; found = data[zone]; next = 1.0;
          } else {
            expected = -1.0; found = data[zone]; next = -1.0;
          }
          if (found != expected) FPRINTF(stdout, "zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
          //FPRINTF(stdout, "%p[%i] = %f\n", data, zone, 1.0);
          data[zone] = next;
        });
      }
      */

      r2.start("wait-recv", Range::pink);
      tm.start("wait-recv");

      comm.waitRecv();

      tm.stop();
      r2.restart("wait-send", Range::pink);
      tm.start("wait-send");

      comm.waitSend();

      tm.stop();
      r2.restart("post-comm", Range::red);
      tm.start("post-comm");

      for (IdxT i = 0; i < num_vars; ++i) {
      
        DataT* data = vars[i].data();
        
        for_all_3d(pol_loop{}, 0, klen,
                               0, jlen,
                               0, ilen,
                               detail::reset_1(ilen, ijlen, data, imin, jmin, kmin, imax, jmax, kmax));
      }
      
      synchronize(pol_loop{});

      tm.stop();
      r2.stop();

    }

    tm.print();
    tm.clear();
}
 
template < typename pol_type >
void prime_allocator(pol_type const& pol, Allocator& aloc, Timer& tm, IdxT num_vars, IdxT len)
{
  DataT** vars = new DataT*[num_vars];
  
  tm.start(aloc.name());
    
  for (IdxT i = 0; i < num_vars; ++i) {
    vars[i] = (DataT*)aloc.allocate(len*sizeof(DataT));
  }
  
  for (IdxT i = 0; i < num_vars; ++i) {
  
    DataT* data = vars[i];
    
    for_all(pol, 0, len, [=] HOST DEVICE (IdxT, IdxT idx) {
      data[idx] = 0.0;
    });
  }
  
  for (IdxT i = 0; i < num_vars; ++i) {
    aloc.deallocate(vars[i]);
  }
  
  tm.stop();
  
  delete[] vars;
}

int main(int argc, char** argv)
{
  int required = MPI_THREAD_SINGLE;
  int provided = detail::MPI::Init_thread(&argc, &argv, required);
  
  CommInfo comminfo;
  
  if (required != provided) {
    comminfo.abort_master("Didn't receive MPI thread support required %i provided %i.\n", required, provided);
  }
  
  comminfo.print_master("Started rank %i of %i\n", comminfo.rank, comminfo.size);

  cudaCheck(hipDeviceSynchronize());  

  IdxT sizes[3] = {0, 0, 0};
  int divisions[3] = {0, 0, 0};
  int periodic[3] = {0, 0, 0};
  IdxT ghost_width = 1;
  IdxT num_vars = 1;
  IdxT ncycles = 5;
  
  IdxT i = 1;
  IdxT s = 0;
  for(; i < argc; ++i) {
    if (argv[i][0] == '-') {
      // options
      if (strcmp(&argv[i][1], "comm") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          ++i;
          if (strcmp(argv[i], "mock") == 0) {
            comminfo.mock_communication = true;
          } else if (strcmp(argv[i], "cutoff") == 0) {
            if (i+1 < argc && argv[i+1][0] != '-') {
              comminfo.cutoff = static_cast<IdxT>(atoll(argv[++i]));
            } else {
              comminfo.warn_master("No argument to sub-option, ignoring %s.\n", argv[i]);
            }
          } else if ( strcmp(argv[i], "post_recv") == 0
                   || strcmp(argv[i], "post_send") == 0
                   || strcmp(argv[i], "wait_recv") == 0
                   || strcmp(argv[i], "wait_send") == 0 ) {
            CommInfo::method* method = nullptr;
            if (strcmp(argv[i], "post_recv") == 0) {
              method = &comminfo.post_recv_method;
            } else if (strcmp(argv[i], "post_send") == 0) {
              method = &comminfo.post_send_method;
            } else if (strcmp(argv[i], "wait_recv") == 0) {
              method = &comminfo.wait_recv_method;
            } else if (strcmp(argv[i], "wait_send") == 0) {
              method = &comminfo.wait_send_method;
            }
            if (i+1 < argc && method != nullptr) {
              ++i;
              if (strcmp(argv[i], "wait_any") == 0) {
                *method = CommInfo::method::waitany;
              } else if (strcmp(argv[i], "wait_some") == 0) {
                *method = CommInfo::method::waitsome;
              } else if (strcmp(argv[i], "wait_all") == 0) {
                *method = CommInfo::method::waitall;
              } else if (strcmp(argv[i], "test_any") == 0) {
                *method = CommInfo::method::testany;
              } else if (strcmp(argv[i], "test_some") == 0) {
                *method = CommInfo::method::testsome;
              } else if (strcmp(argv[i], "test_all") == 0) {
                *method = CommInfo::method::testall;
              } else {
                comminfo.warn_master("Invalid argument to sub-option, ignoring %s.\n", argv[i-1]);
              }
            } else {
              comminfo.warn_master("No argument to sub-option, ignoring %s.\n", argv[i]);
            }
          } else {
            comminfo.warn_master("Invalid argument to option, ignoring %s.\n", argv[i-1]);
          }
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else if (strcmp(&argv[i][1], "ghost") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          ghost_width = static_cast<IdxT>(atoll(argv[++i]));
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else if (strcmp(&argv[i][1], "vars") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          num_vars = static_cast<IdxT>(atoll(argv[++i]));
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else if (strcmp(&argv[i][1], "cycles") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          ncycles = static_cast<IdxT>(atoll(argv[++i]));
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else if (strcmp(&argv[i][1], "periodic") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          int ret = sscanf(argv[++i], "%d_%d_%d", &periodic[0], &periodic[1], &periodic[2]);
          if (ret == 1) {
            periodic[1] = periodic[0];
            periodic[2] = periodic[0];
          } else if (ret != 3) {
            periodic[0] = 0;
            periodic[1] = 0;
            periodic[2] = 0;
            comminfo.warn_master("Invalid arguments to option, ignoring %s.\n", argv[i-1]);
          }
          periodic[0] = periodic[0] ? 1 : 0;
          periodic[1] = periodic[1] ? 1 : 0;
          periodic[2] = periodic[2] ? 1 : 0;
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else if (strcmp(&argv[i][1], "divide") == 0) {
        if (i+1 < argc && argv[i+1][0] != '-') {
          int ret = sscanf(argv[++i], "%d_%d_%d", &divisions[0], &divisions[1], &divisions[2]);
          if (ret != 3 || divisions[0] < 1 || divisions[1] < 1 || divisions[2] < 1) {
            divisions[0] = 0;
            divisions[1] = 0;
            divisions[2] = 0;
            comminfo.warn_master("Invalid arguments to option, ignoring %s.\n", argv[i-1]);
          }
        } else {
          comminfo.warn_master("No argument to option, ignoring %s.\n", argv[i]);
        }
      } else {
        comminfo.warn_master("Unknown option, ignoring %s.\n", argv[i]); 
      }
    } else if (std::isdigit(argv[i][0]) && s < 1) {
      long read_sizes[3] {0, 0, 0};
      int ret = sscanf(argv[i], "%ld_%ld_%ld", &read_sizes[0], &read_sizes[1], &read_sizes[2]);
      if (ret == 1) {
        ++s;
        sizes[0] = read_sizes[0];
        sizes[1] = read_sizes[0];
        sizes[2] = read_sizes[0];
      } else if (ret == 3) {
        ++s;
        sizes[0] = read_sizes[0];
        sizes[1] = read_sizes[1];
        sizes[2] = read_sizes[2];
      } else {
        // set sizes invalid
        sizes[0] = 0;
        sizes[1] = 0;
        sizes[2] = 0;
        comminfo.warn_master("Invalid sizes argument, ignoring %s.\n", argv[i]);
      }
    } else {
      comminfo.warn_master("Invalid argument, ignoring %s.\n", argv[i]);
    }
  }
  
  if (ncycles <= 0) {
    comminfo.abort_master("Invalid cycles argument.\n");
  } else if (num_vars <= 0) {
    comminfo.abort_master("Invalid vars argument.\n");
  } else if (ghost_width <= 0) {
    comminfo.abort_master("Invalid ghost argument.\n");
  } else if ( (divisions[0] != 0 || divisions[1] != 0 || divisions[2] != 0)
           && comminfo.size != divisions[0] * divisions[1] * divisions[2]) {
    comminfo.abort_master("Invalid mesh divisions\n");
  }
  
  GlobalMeshInfo global_info(sizes, comminfo.size, divisions, periodic, ghost_width);
  
  // create cartesian communicator and get rank
  comminfo.cart.create(global_info.divisions, global_info.periodic);
  
  MeshInfo info = MeshInfo::get_local(global_info, comminfo.cart.coords);
  
  // print info about problem setup
  comminfo.print_master("Do %s communication\n", comminfo.mock_communication ? "mock" : "real");
  comminfo.print_master("Message policy cutoff %i\n", comminfo.cutoff);
  comminfo.print_master("Post Recv using %s method\n", CommInfo::method_str(comminfo.post_recv_method));
  comminfo.print_master("Post Send using %s method\n", CommInfo::method_str(comminfo.post_send_method));
  comminfo.print_master("Wait Recv using %s method\n", CommInfo::method_str(comminfo.wait_recv_method));
  comminfo.print_master("Wait Send using %s method\n", CommInfo::method_str(comminfo.wait_send_method));
  comminfo.print_master("Num cycles  %i\n", ncycles);
  comminfo.print_master("Num cycles  %i\n", ncycles);
  comminfo.print_master("Num vars    %i\n", num_vars);
  comminfo.print_master("ghost_width %i\n", info.ghost_width);
  comminfo.print_master("size      %8i %8i %8i\n", global_info.sizes[0],       global_info.sizes[1],       global_info.sizes[2]);
  comminfo.print_master("divisions %8i %8i %8i\n", comminfo.cart.divisions[0], comminfo.cart.divisions[1], comminfo.cart.divisions[2]);
  comminfo.print_master("periodic  %8i %8i %8i\n", comminfo.cart.periodic[0],  comminfo.cart.periodic[1],  comminfo.cart.periodic[2]);
  comminfo.print_master("division map\n", comminfo.cart.periodic[0], comminfo.cart.periodic[1], comminfo.cart.periodic[2]);
  // print division map
  IdxT max_cuts = std::max(std::max(comminfo.cart.divisions[0], comminfo.cart.divisions[1]), comminfo.cart.divisions[2]);
  for (IdxT ci = 0; ci <= max_cuts; ++ci) {
    int division_coords[3] {-1, -1, -1};
    if (ci <= comminfo.cart.divisions[0]) {
      division_coords[0] = ci * (sizes[0] / comminfo.cart.divisions[0]) + std::min(ci, sizes[0] % comminfo.cart.divisions[0]);
    }
    if (ci <= comminfo.cart.divisions[0]) {
      division_coords[1] = ci * (sizes[1] / comminfo.cart.divisions[1]) + std::min(ci, sizes[1] % comminfo.cart.divisions[1]);
    }
    if (ci <= comminfo.cart.divisions[0]) {
      division_coords[2] = ci * (sizes[2] / comminfo.cart.divisions[2]) + std::min(ci, sizes[2] % comminfo.cart.divisions[2]);
    }
    comminfo.print_master("map       %8i %8i %8i\n", division_coords[0], division_coords[1], division_coords[2] );
  }
  
  HostAllocator host_alloc;
  HostPinnedAllocator hostpinned_alloc;
  DeviceAllocator device_alloc;
  ManagedAllocator managed_alloc;
  ManagedHostPreferredAllocator managed_host_preferred_alloc;
  ManagedHostPreferredDeviceAccessedAllocator managed_host_preferred_device_accessed_alloc;
  ManagedDevicePreferredAllocator managed_device_preferred_alloc;
  ManagedDevicePreferredHostAccessedAllocator managed_device_preferred_host_accessed_alloc;
  
  Timer tm(1024);

  // warm-up memory pools
  {
    Range r("Memmory pool init", Range::green);
    
    FPRINTF(stdout, "Starting up memory pools\n");
    
    prime_allocator(seq_pol{},        host_alloc,                                   tm, num_vars+1, info.totallen);
    
    prime_allocator(omp_pol{},        hostpinned_alloc,                             tm, num_vars+1, info.totallen);
    
    prime_allocator(cuda_pol{},       device_alloc,                                 tm, num_vars+1, info.totallen);
    
    prime_allocator(cuda_batch_pol{}, managed_alloc,                                tm, num_vars+1, info.totallen);
    
    prime_allocator(seq_pol{},        managed_host_preferred_alloc,                 tm, num_vars+1, info.totallen);
    
    prime_allocator(omp_pol{},        managed_host_preferred_device_accessed_alloc, tm, num_vars+1, info.totallen);
    
    prime_allocator(cuda_pol{},       managed_device_preferred_alloc,               tm, num_vars+1, info.totallen);
    
    prime_allocator(cuda_batch_pol{}, managed_device_preferred_host_accessed_alloc, tm, num_vars+1, info.totallen);

    tm.print();
    tm.clear();

  }

  // host allocated
  {
    Allocator& mesh_aloc = host_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);
  }

  // host pinned allocated
  {
    Allocator& mesh_aloc = hostpinned_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // device allocated
  {
    Allocator& mesh_aloc = device_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    // do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
    
    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
 
    if (comminfo.mock_communication) {
      do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, device_alloc, device_alloc, tm);
 
      do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, device_alloc, device_alloc, tm);
    }
  }

  // managed allocated
  {
    Allocator& mesh_aloc = managed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed host preferred allocated
  {
    Allocator& mesh_aloc = managed_host_preferred_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed host preferred device accessed allocated
  {
    Allocator& mesh_aloc = managed_host_preferred_device_accessed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed device preferred allocated
  {
    Allocator& mesh_aloc = managed_device_preferred_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed device preferred host accessed allocated
  {
    Allocator& mesh_aloc = managed_device_preferred_host_accessed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);

    do_cycles<cuda_pol, cuda_batch_pol, seq_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, tm);
    
    do_cycles<cuda_pol, cuda_batch_pol, cuda_batch_pol>(comminfo, info, num_vars, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  comminfo.cart.disconnect();
  detail::MPI::Finalize();
  return 0;
}

