#include <cstdio>
#include <cstdlib>

#include <mpi.h>

#include "memory.cuh"
#include "for_all.cuh"
#include "profiling.cuh"
#include "mesh.cuh"
#include "comm.cuh"

namespace detail {

  struct set_n1 {
     DataT* data;
     set_n1(DataT* data_) : data(data_) {}
     HOST DEVICE
     void operator()(IdxT i, IdxT) const {
       IdxT zone = i;
       //printf("%p[%i] = %f\n", data, zone, 1.0); fflush(stdout);
       data[zone] = -1.0;
     }
  };

  struct set_1 {
     IdxT ilen, ijlen;
     DataT* data;
     set_1(IdxT ilen_, IdxT ijlen_, DataT* data_) : ilen(ilen_), ijlen(ijlen_), data(data_) {}
     HOST DEVICE
     void operator()(IdxT k, IdxT j, IdxT i, IdxT idx) const {
       IdxT zone = i + j * ilen + k * ijlen;
       //printf("%p[%i] = %f\n", data, zone, 1.0); fflush(stdout);
       data[zone] = 1.0;
     }
  };

  struct reset_1 {
     IdxT ilen, ijlen;
     DataT* data;
     IdxT imin, jmin, kmin;
     IdxT imax, jmax, kmax;
     reset_1(IdxT ilen_, IdxT ijlen_, DataT* data_, IdxT imin_, IdxT jmin_, IdxT kmin_, IdxT imax_, IdxT jmax_, IdxT kmax_)
       : ilen(ilen_), ijlen(ijlen_), data(data_)
       , imin(imin_), jmin(jmin_), kmin(kmin_)
       , imax(imax_), jmax(jmax_), kmax(kmax_)
     {}
     HOST DEVICE
     void operator()(IdxT k, IdxT j, IdxT i, IdxT idx) const {
       IdxT zone = i + j * ilen + k * ijlen;
       DataT expected, found, next;
       if (k >= kmin && k < kmax &&
           j >= jmin && j < jmax &&
           i >= imin && i < imax) {
         expected = 1.0; found = data[zone]; next = 1.0;
       } else {
         expected = 0.0; found = data[zone]; next = -1.0;
       }
       //if (found != expected) printf("zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
       //printf("%p[%i] = %f\n", data, zone, 1.0); fflush(stdout);
       data[zone] = next;
     }
  };

} // namespace detail

template < typename pol_loop, typename pol_face, typename pol_edge, typename pol_corner >
void do_cycles(MeshInfo& mesh, IdxT ncycles, Allocator& aloc_mesh, Allocator& aloc_face, Allocator& aloc_edge, Allocator& aloc_corner, Timer& tm)
{
    tm.clear();

    char rname[1024] = ""; snprintf(rname, 1024, "Buffers %s %s %s %s %s %s", pol_face::name, aloc_face.name(), pol_edge::name, aloc_edge.name(), pol_corner::name, aloc_corner.name());
    char test_name[1024] = ""; snprintf(test_name, 1024, "Mesh %s %s %s", pol_loop::name, aloc_mesh.name(), rname);
    printf("Starting test %s\n", test_name); fflush(stdout);

    Range r0(rname, Range::orange);

    tm.start("start-up");

    MeshData var(mesh, aloc_mesh);
    Comm<pol_corner, pol_edge, pol_face> comm(var, aloc_face, aloc_edge, aloc_corner);

    {
      var.allocate();
      DataT* data = var.data();
      IdxT ijklen = mesh.ijklen;

      for_all(pol_loop{}, 0, ijklen,
                          detail::set_n1(data));

      if (pol_loop::async) {cudaCheck(hipDeviceSynchronize());}

    }

    tm.stop();

    for(IdxT cycle = 0; cycle < ncycles; cycle++) {

      Range r1("cycle", Range::yellow);

      IdxT imin = mesh.imin;
      IdxT jmin = mesh.jmin;
      IdxT kmin = mesh.kmin;
      IdxT imax = mesh.imax;
      IdxT jmax = mesh.jmax;
      IdxT kmax = mesh.kmax;
      IdxT ilen = mesh.ilen;
      IdxT jlen = mesh.jlen;
      IdxT klen = mesh.klen;
      IdxT ijlen = mesh.ijlen;
      
      DataT* data = var.data();
      
      Range r2("pre-comm", Range::red);
      tm.start("pre-comm");

      for_all_3d(pol_loop{}, kmin, kmax,
                             jmin, jmax,
                             imin, imax,
                             detail::set_1(ilen, ijlen, data));

      if (pol_loop::async) {cudaCheck(hipDeviceSynchronize());}

      tm.stop();
      r2.restart("post-recv", Range::pink);
      tm.start("post-recv");
      
      comm.postRecv();

      tm.stop();
      r2.restart("post-send", Range::pink);
      tm.start("post-send");

      comm.postSend();

      if (pol_corner::async || pol_edge::async || pol_face::async) {cudaCheck(hipDeviceSynchronize());}
      
/*      for_all_3d(pol_loop{}, 0, klen,
                            0, jlen,
                            0, ilen,
                            [=] (IdxT k, IdxT j, IdxT i, IdxT idx) {
        IdxT zone = i + j * ilen + k * ijlen;
        DataT expected, found, next;
        if (k >= kmin && k < kmax &&
            j >= jmin && j < jmax &&
            i >= imin && i < imax) {
          expected = 1.0; found = data[zone]; next = 1.0;
        } else {
          expected = -1.0; found = data[zone]; next = -1.0;
        }
        if (found != expected) printf("zone %i(%i %i %i) = %f expected %f\n", zone, i, j, k, found, expected);
        //printf("%p[%i] = %f\n", data, zone, 1.0); fflush(stdout);
        data[zone] = next;
      });
*/

      tm.stop();
      r2.restart("wait-send", Range::pink);
      tm.start("wait-send");

      comm.waitSend();

      if (pol_corner::async || pol_edge::async || pol_face::async) {cudaCheck(hipDeviceSynchronize());}

      tm.stop();
      r2.restart("wait-recv", Range::pink);
      tm.start("wait-recv");

      comm.waitRecv();

      tm.stop();
      r2.restart("post-comm", Range::red);
      tm.start("post-comm");

      for_all_3d(pol_loop{}, 0, klen,
                             0, jlen,
                             0, ilen,
                             detail::reset_1(ilen, ijlen, data, imin, jmin, kmin, imax, jmax, kmax));

      if (pol_loop::async) {cudaCheck(hipDeviceSynchronize());}

      tm.stop();
      r2.stop();

    }

    tm.print();
    tm.clear();
}
 

int main(int argc, char** argv)
{
  int required = MPI_THREAD_SINGLE;
  int provided = MPI_THREAD_SINGLE;
  MPI_Init_thread(&argc, &argv, required, &provided);

  MPI_Comm mpi_comm = MPI_COMM_WORLD;

  if (required != provided) {
    fprintf(stderr, "Didn't receive MPI thread support required %i provided %i.\n", required, provided); fflush(stderr);
    MPI_Abort(mpi_comm, 1);
  }

  int comm_rank = -1;
  MPI_Comm_rank(mpi_comm, &comm_rank);
  int comm_size = 0;
  MPI_Comm_size(mpi_comm, &comm_size);

  if (comm_rank == 0) {
    printf("Started rank %i of %i\n", comm_rank, comm_size); fflush(stdout);
  }

  cudaCheck(hipDeviceSynchronize());  

  IdxT isize = 0;
  IdxT jsize = 0;
  IdxT ksize = 0;

  if (argc == 1) {
    isize = 100;
    jsize = 100;
    ksize = 100;
  } else if (argc == 2) {
    isize = static_cast<IdxT>(atoll(argv[1]));
    jsize = isize;
    ksize = isize;
  } else if (argc == 4) {
    isize = static_cast<IdxT>(atoll(argv[1]));
    jsize = static_cast<IdxT>(atoll(argv[2]));
    ksize = static_cast<IdxT>(atoll(argv[3]));
  } else {
    if (comm_rank == 0) {
      fprintf(stderr, "Invalid arguments.\n"); fflush(stderr);
    }
    MPI_Abort(MPI_COMM_WORLD, 1);
  }

  if (isize <= 0 || jsize <= 0 || ksize <= 0) {
    if (comm_rank == 0) {
      fprintf(stderr, "Invalid size arguments.\n"); fflush(stderr);
    }
    MPI_Abort(MPI_COMM_WORLD, 1);
  }

  MeshInfo mesh(isize, jsize, ksize);
    
  if (comm_rank == 0) {
    printf("Mesh info\n");
    printf("%i %i %i\n", mesh.isize, mesh.jsize, mesh.ksize);
    printf("ij %i ik %i jk %i\n", mesh.ijsize, mesh.iksize, mesh.jksize);
    printf("ijk %i\n", mesh.ijksize);
    printf("i %8i %8i %8i %8i\n", 0, mesh.imin, mesh.imax, mesh.ilen);
    printf("j %8i %8i %8i %8i\n", 0, mesh.jmin, mesh.jmax, mesh.jlen);
    printf("k %8i %8i %8i %8i\n", 0, mesh.kmin, mesh.kmax, mesh.klen);
    printf("ij %i ik %i jk %i\n", mesh.ijlen, mesh.iklen, mesh.jklen);
    printf("ijk %i\n", mesh.ijklen);
    fflush(stdout);
  }
  
  HostAllocator host_alloc;
  HostPinnedAllocator hostpinned_alloc;
  DeviceAllocator device_alloc;
  ManagedAllocator managed_alloc;
  ManagedHostPreferredAllocator managed_host_preferred_alloc;
  ManagedHostPreferredDeviceAccessedAllocator managed_host_preferred_device_accessed_alloc;
  ManagedDevicePreferredAllocator managed_device_preferred_alloc;
  ManagedDevicePreferredHostAccessedAllocator managed_device_preferred_host_accessed_alloc;
  
  Timer tm(1024);

  // warm-up memory pools
  {
    printf("Starting up memory pools\n"); fflush(stdout);

    Range r("Memmory pool init", Range::green);

    void* var0;
    void* var1;
 
    tm.start(host_alloc.name());

    var0 = host_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = host_alloc.allocate(mesh.ijksize*sizeof(DataT));
    host_alloc.deallocate(var0);
    host_alloc.deallocate(var1);

    tm.restart(hostpinned_alloc.name());

    var0 = hostpinned_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = hostpinned_alloc.allocate(mesh.ijksize*sizeof(DataT));
    hostpinned_alloc.deallocate(var0);
    hostpinned_alloc.deallocate(var1);

    tm.restart(device_alloc.name());

    var0 = device_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = device_alloc.allocate(mesh.ijksize*sizeof(DataT));
    device_alloc.deallocate(var0);
    device_alloc.deallocate(var1);

    tm.restart(managed_alloc.name());

    var0 = managed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = managed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    managed_alloc.deallocate(var0);
    managed_alloc.deallocate(var1);

    tm.restart(managed_host_preferred_alloc.name());

    var0 = managed_host_preferred_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = managed_host_preferred_alloc.allocate(mesh.ijksize*sizeof(DataT));
    managed_host_preferred_alloc.deallocate(var0);
    managed_host_preferred_alloc.deallocate(var1);

    tm.restart(managed_host_preferred_device_accessed_alloc.name());

    var0 = managed_host_preferred_device_accessed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = managed_host_preferred_device_accessed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    managed_host_preferred_device_accessed_alloc.deallocate(var0);
    managed_host_preferred_device_accessed_alloc.deallocate(var1);

    tm.restart(managed_device_preferred_alloc.name());

    var0 = managed_device_preferred_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = managed_device_preferred_alloc.allocate(mesh.ijksize*sizeof(DataT));
    managed_device_preferred_alloc.deallocate(var0);
    managed_device_preferred_alloc.deallocate(var1);

    tm.restart(managed_device_preferred_alloc.name());

    var0 = managed_device_preferred_host_accessed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    var1 = managed_device_preferred_host_accessed_alloc.allocate(mesh.ijksize*sizeof(DataT));
    managed_device_preferred_host_accessed_alloc.deallocate(var0);
    managed_device_preferred_host_accessed_alloc.deallocate(var1);

    tm.stop();

    tm.print();
    tm.clear();

  }

  IdxT ncycles = 5;

  // host allocated
  {
    Allocator& mesh_aloc = host_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);
  }

  // host pinned allocated
  {
    Allocator& mesh_aloc = hostpinned_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // device allocated
  {
    Allocator& mesh_aloc = device_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    // do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    // do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
 
    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, device_alloc, device_alloc, device_alloc, tm);
  }

  // managed allocated
  {
    Allocator& mesh_aloc = managed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed host preferred allocated
  {
    Allocator& mesh_aloc = managed_host_preferred_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed host preferred device accessed allocated
  {
    Allocator& mesh_aloc = managed_host_preferred_device_accessed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed device preferred allocated
  {
    Allocator& mesh_aloc = managed_device_preferred_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  // managed device preferred host accessed allocated
  {
    Allocator& mesh_aloc = managed_device_preferred_host_accessed_alloc;

    char name[1024] = ""; snprintf(name, 1024, "Mesh %s", mesh_aloc.name());
    Range r0(name, Range::blue);

    do_cycles<seq_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, seq_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, host_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, seq_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, host_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, seq_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, host_alloc, tm);

    do_cycles<cuda_pol, cuda_pol, cuda_pol, cuda_pol>(mesh, ncycles, mesh_aloc, hostpinned_alloc, hostpinned_alloc, hostpinned_alloc, tm);
  }

  MPI_Finalize();
  return 0;
}

